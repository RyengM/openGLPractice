#include "hip/hip_runtime.h"
#include <headFiles/Smoke.h>

#include <cuda_gl_interop.h>
#include <iostream>

// cuda_gl_interop.h has included hip/hip_runtime_api.h and gl.h

__global__ void test_kernel()
{

}

void crazy()
{
    test_kernel<<<1, 1>>>();   
    std::cout << "Hello World!\n" << std::endl;  
}