
#include <hip/hip_runtime.h>
//#include "headfiles/smoke.h"
//#include "headfiles/cudautilities.hpp"
//
//#include <cuda.h>
//#include <cuda_runtime_api.h>
//#include <device_launch_parameters.h>
//#include <cuda_gl_interop.h>
//
//static __global__ void init_curand_kernel(curandstate* state, size_t block_size)
//{
//    size_t i = threadidx.x;
//    curand_init(0, i, 0, &state[i]);
//}
//
//static __global__ void init_kernel(curandstate* curand_state, particles* particles)
//{
//    int index = threadidx.x;
//    particles[index].active = false;
//    particles[index].offset = glm::vec3(0, 0, 0);
//    particles[index].velocity = glm::vec3(&curand_state[index], &curand_state[index], &curand_state[index]);
//}
//
//static __global__ void update_kernel(particles* particles)
//{
//    int index = threadidx.x;
//    if (particles[index].active)
//        particles[index].offset += particles[index].velocity;
//}
//
//void smoke::init_cuda()
//{
//    checkcudaerrors(cudasetdevice(0));
//
//    checkcudaerrors(cudagraphicsglregisterbuffer(&position_buffer_resource_, position_buffer_, cudagraphicsmapflagswritediscard));
//    checkcudaerrors(cudamalloc((void**)&curand_state, max_particles * sizeof(curandstate)));
//
//    init_curand_kernel<<<1, max_particles>>>(curand_state, max_particles);
//
//    init_kernel<<<1, max_particles>>>(curand_state, particles);
//}
//
//void smoke::update_cuda()
//{
//    float4* position_buffer;
//
//    checkcudaerrors(cudasetdevice(0));
//
//    checkcudaerrors(cudagraphicsmapresources(1, &position_buffer_resource_, null));
//    checkcudaerrors(cudagraphicsresourcegetmappedpointer((void **)&position_buffer, nullptr, position_buffer_resource_));
//
//    update_kernel<<<1, max_particles>>>(particles);
//}
//
//void smoke::free_cuda()
//{
//    checkcudaerrors(cudasetdevice(0));
//    checkcudaerrors(cudafree(curand_state));
//    checkcudaerrors(cudagraphicsunregisterresource(position_buffer_resource_));
//}
