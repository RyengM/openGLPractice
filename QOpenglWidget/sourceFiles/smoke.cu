
#include <hip/hip_runtime.h>
//#include "headfiles/smoke.h"
//#include "headfiles/cudautilities.hpp"
//
//#include <cuda.h>
//#include <cuda_runtime_api.h>
//#include <device_launch_parameters.h>
//#include <cuda_gl_interop.h>
//
//static __global__ void init_curand_kernel(curandState* state, unsigned int block_size)
//{
//    unsigned int i = threadIdx.x;
//    curand_init(0, i, 0, &state[i]);
//}
//
//static __global__ void init_kernel(curandState* curand_state, Particles* particles)
//{
//    int index = threadIdx.x;
//    particles[index].active = false;
//    particles[index].offset = glm::vec3(0, 0, 0);
//    particles[index].velocity = glm::vec3(&curand_state[index], &curand_state[index], &curand_state[index]);
//}
//
//static __global__ void update_kernel(Particles* particles)
//{
//    int index = threadIdx.x;
//    if (particles[index].active)
//        particles[index].offset += particles[index].velocity;
//}
//
//void Smoke::init_cuda()
//{
//    checkCudaErrors(cudaSetDevice(0));
//    
//    checkCudaErrors(cudaGraphicsGLRegisterBuffer(&position_buffer_resource_, position_buffer_, cudaGraphicsMapFlagsWriteDiscard));
//    checkCudaErrors(cudaMalloc((void**)&curand_state, MAX_PARTICLES * sizeof(curandState)));
//
//    init_curand_kernel<<<1, MAX_PARTICLES>>>(curand_state, MAX_PARTICLES);
//
//    init_kernel<<<1, MAX_PARTICLES>>>(curand_state, particles);
//}
//
//void Smoke::update_cuda()
//{
//    float4* position_buffer;
//
//    checkCudaErrors(cudaSetDevice(0));
//
//    checkCudaErrors(cudaGraphicsMapResources(1, &position_buffer_resource_));
//    checkCudaErrors(cudaGraphicsResourceGetMappedPointer((void **)&position_buffer, nullptr, position_buffer_resource_));
//
//    update_kernel<<<1, MAX_PARTICLES>>>(particles);
//}
//
//void Smoke::free_cuda()
//{
//    checkCudaErrors(cudaSetDevice(0));
//    checkCudaErrors(cudaFree(curand_state));
//    checkCudaErrors(cudaGraphicsUnregisterResource(position_buffer_resource_));
//}
