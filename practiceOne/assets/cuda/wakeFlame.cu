#include "hip/hip_runtime.h"
#include <../assets/cuda/wakeFlame.cuh>
#include <myTools/currentWakeFlameData.h>
#include <myTools/cudaUtility.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <glm/gtc/type_ptr.hpp>
#include <glm/gtc/matrix_transform.hpp>

#include <myTools/cudaUtility.h>

hiprandState_t* states;

__global__ void init_curand_state(unsigned int seed, hiprandState_t* states) {
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int size = int(ceil(sqrtf(float(PARTICLE_NUM))));

	if (row > size - 1 || col > size - 1)
		return;

	auto i = col + row * size;
	hiprand_init(seed, i, 0, &states[i]);
}

void initialize_flame(CurrentWakeFlameData::wakeFlameParticle* _flame,
	CurrentWakeFlameData::CurrentData& _current)
{
	int size = int(ceil(sqrtf(PARTICLE_NUM)));
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size(static_cast<unsigned int>(ceil(size / float(block_size.x))),
		static_cast<unsigned int>(ceil(size / float(block_size.y))));

	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&states), PARTICLE_NUM * sizeof(hiprandState_t)));
	init_curand_state<<<grid_size, block_size>>>(time(nullptr), states);

	glm::mat4 translate_to_mtx = translate(glm::mat4(1.f), -_current.position);
	glm::mat4 rotate_mtx = rotate(glm::mat4(1.f), glm::radians(-_current.angle.x), glm::vec3(0.f, 1.f, 0.f));
	glm::mat4 translate_back_mtx = translate(glm::mat4(1.f), _current.position);
	glm::mat4 transform_mtx = translate_back_mtx * rotate_mtx * translate_to_mtx;

	initialize_flame_kernel<<<grid_size, block_size>>>(_flame, _current, size, states, transform_mtx);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

__global__ void initialize_flame_kernel(CurrentWakeFlameData::wakeFlameParticle* _flame,
	CurrentWakeFlameData::CurrentData _current,int size, hiprandState_t* states, glm::mat4 transform)
{
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;

	if (row > size - 1 || col > size - 1)
		return;
	
	int i = col + row * size;
	initialize_single_particle(_flame, i, _current, states[i], transform);
}

__device__ void initialize_single_particle(CurrentWakeFlameData::wakeFlameParticle* _flame, int index,
	CurrentWakeFlameData::CurrentData _current,hiprandState_t state, glm::mat4 transform)
{
	glm::vec3 position(hiprand_uniform(&state), hiprand_uniform(&state), hiprand_uniform(&state));
	position = transform * glm::vec4(position, 1.0f);
	
	_flame[index].active = true;
	_flame[index].x = position.x;
	_flame[index].y = position.y;
	_flame[index].z = position.z;
	_flame[index].r = hiprand_uniform(&state);
	_flame[index].g = hiprand_uniform(&state);
	_flame[index].b = hiprand_uniform(&state);
	_flame[index].dim = 0.5;
}

void update_flame(CurrentWakeFlameData::wakeFlameParticle* _flame, CurrentWakeFlameData::CurrentData& _current,
			float* verticesArray) {
	int size = int(ceil(sqrtf(PARTICLE_NUM)));
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size(static_cast<unsigned int>(ceil(size / float(block_size.x))),
		static_cast<unsigned int>(ceil(size / float(block_size.y))));

	glm::mat4 translate_to_mtx = translate(glm::mat4(1.f), -_current.position);
	glm::mat4 rotate_mtx = rotate(glm::mat4(1.f), glm::radians(-_current.angle.x), glm::vec3(0.f, 1.f, 0.f));
	glm::mat4 translate_back_mtx = translate(glm::mat4(1.f), _current.position);
	glm::mat4 transform_mtx = translate_back_mtx * rotate_mtx * translate_to_mtx;

	printf("WTF???\n");
	update_flame_kernel<<<grid_size, block_size>>>(_flame, _current, verticesArray, size, states, transform_mtx);
}

__global__ void update_flame_kernel(CurrentWakeFlameData::wakeFlameParticle* _flame,
	CurrentWakeFlameData::CurrentData _current, float* verticesArray, int size,
    hiprandState_t* states, glm::mat4 transform)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    int i = col + row * size;
	printf("(%d)", i);
    update_single_particle(_flame, i, _current, verticesArray, states[i], transform);
}

__device__ void update_single_particle(CurrentWakeFlameData::wakeFlameParticle* _flame, int index,
	CurrentWakeFlameData::CurrentData current,float* verticesArray,
	hiprandState_t state, glm::mat4 transform)
{
	printf("(%f,%f,%f)", _flame[index].x, _flame[index].y, _flame[index].z);
	glm::vec3 position(hiprand_uniform(&state), hiprand_uniform(&state), hiprand_uniform(&state));
	//position = transform * glm::vec4(position, 1.0f);
	_flame[index].active = true;
	_flame[index].x = position.x;
	_flame[index].y = position.y;
	_flame[index].z = position.z;
	_flame[index].r = hiprand_uniform(&state);
	_flame[index].g = hiprand_uniform(&state);
	_flame[index].b = hiprand_uniform(&state);
	_flame[index].dim = 0.5;

	update_verticesArray(_flame, verticesArray, index);
}

__device__ void update_verticesArray(CurrentWakeFlameData::wakeFlameParticle* _flame,
	float* verticesArray, int index)
{

	printf("(%f,%f,%f)", _flame[index].x, _flame[index].y, _flame[index].z);
	verticesArray[9 * index + 0] = _flame[index].x;
	verticesArray[9 * index + 1] = _flame[index].y;
	verticesArray[9 * index + 2] = _flame[index].z;

	verticesArray[9 * index + 3] = _flame[index].r / 1;
	verticesArray[9 * index + 4] = _flame[index].g / 2;
	verticesArray[9 * index + 5] = _flame[index].b / 3;

	verticesArray[9 * index + 6] = 0.f;
	verticesArray[9 * index + 7] = 0.f;

	verticesArray[9 * index + 8] = _flame[index].dim;
}
